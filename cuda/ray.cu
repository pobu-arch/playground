#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <memory>
#include <vector>
#include <utility>
#include <cstdint>
#include <iostream>
#include <fstream>
#include <cmath>
#include <limits>
#include <random>
#include "hip/hip_runtime.h"
#include ""
#include "geometry.h"
#include "my_kernel.h"
using namespace std;

#define OBJ_NUM 256
#define SCREEN_WIDTH 3600
#define SCREEN_HEIGHT 3600

//#define GPU
#ifdef GPU
    #define BlockDim 32
    #define GridDim  (SCREEN_WIDTH / BlockDim + 1)
#endif

struct coordinate
{
    int x;
    int y;
};

const float kInfinity = std::numeric_limits<float>::max();
random_device rd;
mt19937 gen(rd());
uniform_real_distribution<> dis(0, 1);

inline
__host__ __device__ float clamp(const float& lo, const float& hi, const float& v)
{
    return std::max(lo, std::min(hi, v));
}

inline
__host__ __device__ float deg2rad(const float& deg)
{
    return deg * 3.1415926 / 180;
}

inline
__host__ __device__ Vec3f mix(const Vec3f& a, const Vec3f& b, const float& mixValue)
{
    return a * (1 - mixValue) + b * mixValue;
}

struct Options
{
    uint32_t width;
    uint32_t height;
    float fov;
    Matrix44f cameraToWorld;
};

bool __host__ __device__ solveQuadratic(const float& a, const float& b, const float& c, float& x0, float& x1)
{
    float discr = b * b - 4 * a * c;
    if (discr < 0) return false;
    else if (discr == 0)
    {
        x0 = x1 = -0.5 * b / a;
    }
    else
    {
        float q = (b > 0) ?
            -0.5 * (b + sqrt(discr)) :
            -0.5 * (b - sqrt(discr));
        x0 = q / a;
        x1 = c / q;
    }

    return true;
}

class Sphere
{
public:
    Vec3f color;

    __host__ __device__ Sphere(const Vec3f& c, const float& r) : radius(r), radius2(r* r), center(c) {}

    // ray origin, ray direction, out is the distance from the ray origin to the intersection point
    bool __host__ __device__ intersect(const Vec3f& orig, const Vec3f& dir, float& t) const
    {
        float t0, t1; // solutions for t if the ray intersects
#if 0
        // geometric solution
        Vec3f L = center - orig;
        float tca = L.dotProduct(dir);
        if (tca < 0) return false;
        float d2 = L.dotProduct(L) - tca * tca;
        if (d2 > radius2) return false;
        float thc = sqrt(radius2 - d2);
        t0 = tca - thc;
        t1 = tca + thc;
#else
        // analytic solution
        Vec3f L = orig - center;
        float a = dir.dotProduct(dir);
        float b = 2 * dir.dotProduct(L);
        float c = L.dotProduct(L) - radius2;
        if (!solveQuadratic(a, b, c, t0, t1)) return false;
#endif
        if (t0 > t1) std::swap(t0, t1);

        if (t0 < 0)
        {
            t0 = t1; // if t0 is negative, let's use t1 instead
            if (t0 < 0) return false; // both t0 and t1 are negative
        }

        t = t0;

        return true;
    }
    // [comment]
    // Set surface data such as normal and texture coordinates at a given point on the surface
    //
    // \param Phit is the point ont the surface we want to get data on
    //
    // \param[out] Nhit is the normal at Phit
    //
    // \param[out] tex are the texture coordinates at Phit
    //
    // [/comment]
    void __host__ __device__ getSurfaceData(const Vec3f& Phit, Vec3f& Nhit, Vec2f& tex) const
    {
        Nhit = Phit - center;
        Nhit.normalize();
        // In this particular case, the normal is simular to a point on a unit sphere
        // centred around the origin. We can thus use the normal coordinates to compute
        // the spherical coordinates of Phit.
        // atan2 returns a value in the range [-pi, pi] and we need to remap it to range [0, 1]
        // acosf returns a value in the range [0, pi] and we also need to remap it to the range [0, 1]
        tex.x = (1 + atan2(Nhit.z, Nhit.x) / 3.1415926) * 0.5;
        tex.y = acosf(Nhit.y) / 3.1415926;
    }
    float radius, radius2;
    Vec3f center;
};

// [comment]
// Returns true if the ray intersects an Sphere. The variable tNear is set to the closest intersection distance and hitSphere
// is a pointer to the intersected Sphere. The variable tNear is set to infinity and hitSphere is set null if no intersection
// was found.
// [/comment]
bool trace(const Vec3f& orig, const Vec3f& dir, const std::vector<std::unique_ptr<Sphere>>& Spheres, float& tNear, const Sphere*& hitSphere)
{
    tNear = kInfinity;
    std::vector<std::unique_ptr<Sphere>>::const_iterator iter = Spheres.begin();
    for (; iter != Spheres.end(); ++iter)
    {
        float t = kInfinity;
        if ((*iter)->intersect(orig, dir, t) && t < tNear)
        {
            hitSphere = iter->get();
            tNear = t;
        }
    }

    return (hitSphere != nullptr);
}

// [comment]
// Compute the color at the intersection point if any (returns background color otherwise)
// [/comment]
Vec3f castRay(
    const Vec3f& orig, const Vec3f& dir,
    const std::vector<std::unique_ptr<Sphere>>& Spheres)
{
    Vec3f hitColor = 0;
    const Sphere* hitSphere = nullptr; // this is a pointer to the hit Sphere
    float t; // this is the intersection distance from the ray origin to the hit point
    if (trace(orig, dir, Spheres, t, hitSphere))
    {
        Vec3f Phit = orig + dir * t;
        Vec3f Nhit;
        Vec2f tex;
        hitSphere->getSurfaceData(Phit, Nhit, tex);
        // Use the normal and texture coordinates to shade the hit point.
        // The normal is used to compute a simple facing ratio and the texture coordinate
        // to compute a basic checker board pattern
        float scale = 4;
        float pattern = (fmodf(tex.x * scale, 1) > 0.5) ^ (fmodf(tex.y * scale, 1) > 0.5);
        hitColor = std::max(0.f, Nhit.dotProduct(-dir)) * mix(hitSphere->color, hitSphere->color * 0.8, pattern);
    }

    return hitColor;
}

// [comment]
// The main render function. This where we iterate over all pixels in the image, generate
// primary rays and cast these rays into the scene. The content of the framebuffer is
// saved to a file.
// [/comment]
void render(
    const Options& options,
    const std::vector<std::unique_ptr<Sphere>>& Spheres)
{
    #ifdef GPU

    int Block_Dim = BlockDim;
    int Grid_Dim = GridDim;

    if (Block_Dim * Block_Dim > 1024)
    {
        cout << "[Error] too many threads in block" << endl;
        exit(-1);
    }

    if (Grid_Dim * Block_Dim < options.width)
    {
        cout << "[Error] number of threads in x/y dimensions less than number of array elements" << endl;
        exit(-1);
    }

    cout << "Screen Dimension (x, y) = (" << options.width << ", " << options.height << ")" << endl;

    float* dev_x, * dev_y;
    float* host_x, * host_y;
    int gpu_count;

    hipError_t errorcode = hipGetDeviceCount(&gpu_count);
    if (errorcode == hipErrorNoDevice)
    {
        cout << "[Error] No GPUs are visible" << endl;
        exit(-1);
    }
    else hipSetDevice(0);

    errorcode = hipMalloc((void**)&dev_x, options.width * options.height * sizeof(float)); // allocate memory on device
    if (errorcode != hipSuccess)
    {
        cout << "[Error] Not enough GPU memory for dev_x" << endl;
    }
    errorcode = hipMalloc((void**)&dev_y, options.width * options.height * sizeof(float));
    if (errorcode != hipSuccess)
    {
        cout << "[Error] Not enough GPU memory for dev_y" << endl;
    }

    cout << "Begin GPU computation on dev_x and dev_y" << endl;
    dim3 DimGrid(ceil(options.width / BlockDim), ceil(options.height / BlockDim), 1);
    dim3 DimBlock(BlockDim, BlockDim, 1);
    
    float scale = tan(deg2rad(options.fov * 0.5));
    float imageAspectRatio = options.width / (float)options.height;
    
    Vec3f orig;
    options.cameraToWorld.multVecMatrix(Vec3f(0), orig);
    Vec3f* framebuffer = new Vec3f[options.width * options.height];
    
    gpu_calc_xy <<<DimGrid, DimBlock>>> (dev_x, dev_y, options.width, options.height, scale, imageAspectRatio);
    cout << "Complete GPU computation on dev_x and dev_y" << endl;

    host_x = new float[options.width * options.height]; // dynamically allocated memory for arrays on host
    host_y = new float[options.width * options.height];

    hipMemcpy(host_x, dev_x, options.width * options.height * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(host_y, dev_y, options.width * options.height * sizeof(float), hipMemcpyDeviceToHost);
    cout << "Complete GPU results copyback" << endl;

    for (uint32_t j = 0; j < options.height; ++j)
    {
        for (uint32_t i = 0; i < options.width; ++i)
        {
#ifdef MAYA_STYLE
            float x = (2 * (i + 0.5) / (float)options.width - 1) * scale;
            float y = (1 - 2 * (j + 0.5) / (float)options.height) * scale * 1 / imageAspectRatio;
#elif

            float x = (2 * (i + 0.5) / (float)options.width - 1) * imageAspectRatio * scale;
            float y = (1 - 2 * (j + 0.5) / (float)options.height) * scale;
#endif
            if (x != host_x[j * options.width + i])
            {
                cout << "x[" << j << "]" << "[" << i << "]" << " = " << host_x[j * options.width + i] << ", which should be " << x << endl;
                exit(-1);
            }
            if (y != host_y[j * options.width + i])
            {
                cout << "y[" << j << "]" << "[" << i << "]" << " = " << host_y[j * options.width + i] << ", which should be " << y << endl;
                exit(-1);
            }

            Vec3f dir;
            options.cameraToWorld.multDirMatrix(Vec3f(host_x[j * options.width + i], host_y[j * options.width + i], -1), dir);
            dir.normalize();
            framebuffer[j * options.width + i] = castRay(orig, dir, Spheres);
        }
    }
    #else
    Vec3f* framebuffer = new Vec3f[options.width * options.height];
    Vec3f* pix = framebuffer;
    float scale = tan(deg2rad(options.fov * 0.5));
    float imageAspectRatio = options.width / (float)options.height;
    Vec3f orig;
    options.cameraToWorld.multVecMatrix(Vec3f(0), orig);
    for (uint32_t j = 0; j < options.height; ++j)
    {
        for (uint32_t i = 0; i < options.width; ++i)
        {
            // [comment]
            // Generate primary ray direction. Compute the x and y position
            // of the ray in screen space. This gives a point on the image plane
            // at z=1. From there, we simply compute the direction by normalized
            // the resulting vec3f variable. This is similar to taking the vector
            // between the point on the image plane and the camera origin, which
            // in camera space is (0,0,0):
            //
            // ray.dir = normalize(Vec3f(x,y,-1) - Vec3f(0));
            // [/comment]
#ifdef MAYA_STYLE
            float x = (2 * (i + 0.5) / (float)options.width - 1) * scale;
            float y = (1 - 2 * (j + 0.5) / (float)options.height) * scale * 1 / imageAspectRatio;
#elif

            float x = (2 * (i + 0.5) / (float)options.width - 1) * imageAspectRatio * scale;
            float y = (1 - 2 * (j + 0.5) / (float)options.height) * scale;
#endif
            // [comment]
            // Don't forget to transform the ray direction using the camera-to-world matrix.
            // [/comment]
            Vec3f dir;
            options.cameraToWorld.multDirMatrix(Vec3f(x, y, -1), dir);
            dir.normalize();
            *(pix++) = castRay(orig, dir, Spheres);
        }
    }
#endif

    // Save result to a PPM image (keep these flags if you compile under Windows)
    std::ofstream ofs("./out.ppm", std::ios::out | std::ios::binary);
    ofs << "P6\n" << options.width << " " << options.height << "\n255\n";
    for (uint32_t i = 0; i < options.height * options.width; ++i)
    {
        char r = (char)(255 * clamp(0, 1, framebuffer[i].x));
        char g = (char)(255 * clamp(0, 1, framebuffer[i].y));
        char b = (char)(255 * clamp(0, 1, framebuffer[i].z));
        ofs << r << g << b;
    }

    ofs.close();

    delete[] framebuffer;
}

// [comment]
// In the main function of the program, we create the scene (create Spheres)
// as well as set the options for the render (image widht and height etc.).
// We then call the render function().
// [/comment]
int main(int argc, char** argv)
{
    // creating the scene (adding Spheres and lights)
    std::vector<std::unique_ptr<Sphere>> Spheres;
    
    // setting up options
    Options options;
    uint32_t numSpheres     = OBJ_NUM;
    options.width           = SCREEN_WIDTH;
    options.height          = SCREEN_HEIGHT;
    options.fov = 51.52;
    options.cameraToWorld = Matrix44f(0.945519, 0, -0.325569, 0, -0.179534, 0.834209, -0.521403, 0, 0.271593, 0.551447, 0.78876, 0, 4.208271, 8.374532, 17.932925, 1);

    // generate a scene made of random spheres
    
    gen.seed(0);
    for (uint32_t i = 0; i < numSpheres; ++i)
    {
        Vec3f randPos((0.5 - dis(gen)) * 10, (0.5 - dis(gen)) * 10, (0.5 + dis(gen) * 10));
        float randRadius = (0.5 + dis(gen) * 0.5);
        Spheres.push_back(std::unique_ptr<Sphere>(new Sphere(randPos, randRadius)));
    }

    // finally, render
    render(options, Spheres);

    return 0;
}