#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>



#define NO_CMD_INPUT
#ifdef NO_CMD_INPUT
    #define M        1024
    #define N        1024
    #define P        1024
    #define BlockDim 32
    #define GridDim  (N / BlockDim + 1)
    #define TILE_WIDTH 16
    #define ADJ_FACTOR 3
#endif

#define CPU_PART
//#define GPU_PART
#define CHECK_PART
//#define PRINT_PART

#ifdef GPU_PART
#define GPU_Task1
//#define GPU_Task2
//#define GPU_Task34
#endif

#define rand_max 7
#define FP float

void print_matrix(FP* a, int m, int n)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%f ", a[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void init_matrix(FP* a, int m, int n)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            a[i * n + j] = (FP)rand() / (FP)rand_max;
            //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
        }
    }
}

__global__ void gpu_kernel_task1(FP* a, FP* b, FP* c, int m, int n, int p)
{
    float sum = 0.0f;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < p)
    {
        for (int i = 0; i < n; ++i)
        {
            sum += a[row * n + i] * b[i * p + col];
        }
        c[row * p + col] = sum;
    }
}

__global__ void gpu_kernel_task2(FP* a, FP* b, FP* c, int m, int n, int p)
{
    __shared__ float sharedA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sharedB[TILE_WIDTH][TILE_WIDTH];
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;
    float v = 0.0;

    for (int i = 0; i < (int)(ceil((float)n / TILE_WIDTH)); i++)
    {
        if (i * TILE_WIDTH + tx < n && row < m)
            sharedA[ty][tx] = a[row * n + i * TILE_WIDTH + tx];
        else
            sharedA[ty][tx] = 0.0;

        if (i * TILE_WIDTH + ty < n && col < p)
            sharedB[ty][tx] = b[(i * TILE_WIDTH + ty) * p + col];
        else
            sharedB[ty][tx] = 0.0;
        __syncthreads();

        for (int j = 0; j < TILE_WIDTH; j++)
            v += sharedA[ty][j] * sharedB[j][tx];
        __syncthreads();
    }

    if (row < m && col < p)
        c[row * p + col] = v;
}

__global__ void gpu_kernel_task34(FP* a, FP* b, FP* c, int m, int n, int p)
{
    __shared__ float sharedA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sharedB[TILE_WIDTH][ADJ_FACTOR * TILE_WIDTH];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;
    float v1 = 0.0, v2 = 0.0;

    for (int i = 0; i < (int)(ceil((float)n / TILE_WIDTH)); i++)
    {
        if (i * TILE_WIDTH + tx < n && row < m)
            sharedA[ty][tx] = a[row * n + i * TILE_WIDTH + tx];
        else
            sharedA[ty][tx] = 0.0;

        if (i * TILE_WIDTH + ty < n && col < p)
        {
            sharedB[ty][tx] = b[(i * TILE_WIDTH + ty) * p + col];
            if (col + p / 2 < p)
            {
                sharedB[ty][tx + TILE_WIDTH] = b[(i * TILE_WIDTH + ty) * p + col + p / 2];
            }
        }
        else
            sharedB[ty][tx] = 0.0;
        __syncthreads();

        for (int j = 0; j < TILE_WIDTH; j++)
        {
            v1 += sharedA[ty][j] * sharedB[j][tx];
            v2 += sharedA[ty][j] * sharedB[j][tx + TILE_WIDTH];
        }
        __syncthreads();
    }

    if (row < m && col < p)
    {
        c[row * p + col] = v1;
        if (col + p / 2 < p)
        {
            c[row * p + col + p / 2] = v2;
        }
    }
}

void kij_matrixmult(FP* a, FP* b, FP* kij, int m, int n, int p)
{
    for (int k = 0; k < n; k++)
    {
        for (int i = 0; i < m; i++)
        {
            FP r = a[i * n + k];
            for (int j = 0; j < p; j++)
            {
                kij[i * p + j] += r * b[k * p + j];
            }
        }
    }
}

void check_matrix(FP* a, FP* b, FP* gpu, int m, int n, int p)
{
    FP* kij = (FP*)malloc(m * p * sizeof(FP)); // results from CPU
    memset(kij, 0, m * p * sizeof(FP));

    kij_matrixmult(a, b, kij, m, n, p);

    // diff on result matrix
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < p; j++)
        {
            gpu[i * p + j] -= kij[i * p + j];
        }
    }

    double error, suma, sumb, sumc;
    suma = 0.; sumb = 0; sumc = 0;
    
    // suma, m * n
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            suma += a[i * n + j] * a[i * n + j];
        }
    }

    // sumb, n * p
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < p; j++)
        {
            sumb += b[i * p + j] * b[i * p + j];
        }
    }
    
    // sumc, m * p
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < p; j++)
        {
            sumc += gpu[i * p + j] * gpu[i * p + j];
        }
    }

    suma = sqrt(suma);
    sumb = sqrt(sumb);
    sumc = sqrt(sumc);
    error = sumc / (sqrt(m*p) * suma * sumb);
    printf("Scaled error between GPU and CPU: %f\n", error);
}

int main(int argc, char* argv[])
{
    int Grid_Dim = 1; //Grid dimension, x and y, square
    int Block_Dim = 1; //Block dimension, x and y, square
    int m, n, p; // matrix dimension
    FP* a, *b, *gpu;

    hipEvent_t start, stop; // using cuda events to measure time
    float elapsed_time_ms; // which is applicable for asynchronous code also
    hipError_t errorcode;

#ifdef NO_CMD_INPUT    
    m = M;
    n = N;
    p = P;
    Block_Dim = BlockDim;
    Grid_Dim = GridDim;
#else
    if (argc != 4)
    {
        printf("Usage: matmul <matrix dim> <block dim> <grid dim>\n");
        exit(-1);
    }
    n = atoi(argv[1]);
    Block_Dim = atoi(argv[2]); // Square block
    Grid_Dim = atoi(argv[3]); // Square grid
#endif

    a = (FP*)malloc(m * n * sizeof(FP)); // dynamically allocated memory for arrays on host
    b = (FP*)malloc(n * p * sizeof(FP));
    gpu = (FP*)malloc(m * p * sizeof(FP)); // results from GPU

    FP* kij = (FP*)malloc(m * p * sizeof(FP)); // results from CPU
    memset(kij, 0, m * p * sizeof(FP));

    srand(12345);
    init_matrix(a, m, n);
    init_matrix(b, n, p);

    hipEventCreate(&start); // instrument code to measure start time
    hipEventCreate(&stop);

#ifdef GPU_PART
    int gpucount = 0; // Count of available GPUs
    int gpunum = 0; // Device number to use
    FP* dev_a, * dev_b, * dev_c;

    // --------------------SET PARAMETERS AND DATA -----------------------

    errorcode = cudaGetDeviceCount(&gpucount);
    if (errorcode == cudaErrorNoDevice)
    {
        printf("No GPUs are visible\n");
        exit(-1);
    }
    else
    {
        printf("Device count = %d\n", gpucount);
    }

    if (Block_Dim * Block_Dim > 1024)
    {
        printf("Error, too many threads in block\n");
        exit(-1);
    }
    
    if (Grid_Dim * Block_Dim < n)
    {
        printf("Error, number of threads in x/y dimensions less than number of array elements\n");
        exit(-1);
    }

    cudaSetDevice(gpunum);
    printf("Using device %d\n", gpunum);

    cudaMalloc((void**)&dev_a, m * n * sizeof(FP)); // allocate memory on device
    cudaMalloc((void**)&dev_b, n * p * sizeof(FP));
    cudaMalloc((void**)&dev_c, m * p * sizeof(FP));

    cudaMemcpy(dev_a, a, m * n * sizeof(FP), cudaMemcpyHostToDevice);
    cudaMemcpy(dev_b, b, n * p * sizeof(FP), cudaMemcpyHostToDevice);

    printf("Matrix Dimension (m, n, p) = (%d, %d, %d)\n", m, n, p);

    cudaEventRecord(start, 0);
    // cudaEventSynchronize(start); // not needed

#ifdef GPU_Task1
    printf("Using gpu_kernel_task1 \n");
    dim3 DimGrid((p - 1) / BlockDim + 1, (m - 1) / BlockDim + 1, 1);
    dim3 DimBlock(BlockDim, BlockDim, 1);
    gpu_kernel_task1 <<<DimGrid, DimBlock >>> (dev_a, dev_b, dev_c, m, n, p);
#endif    

#ifdef GPU_Task2
    printf("Using gpu_kernel_task2 \n");
    dim3 DimGrid(ceil(p / BlockDim), ceil(m / BlockDim), 1);
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    gpu_kernel_task2 <<< DimGrid, DimBlock >>> (dev_a, dev_b, dev_c, m, n, p);
#endif

#ifdef GPU_Task34
    printf("Using gpu_kernel_task34 \n");
    dim3 DimGrid(ceil(p / BlockDim / ADJ_FACTOR ), ceil(m / BlockDim), 1);
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    gpu_kernel_task34 <<< DimGrid, DimBlock >>> (dev_a, dev_b, dev_c, m, n, p);
#endif

    cudaEventRecord(stop, 0); // instrument code to measure end time
    cudaEventSynchronize(stop);
    cudaEventElapsedTime(&elapsed_time_ms, start, stop);

    cudaMemcpy(gpu, dev_c, m * p * sizeof(FP), cudaMemcpyDeviceToHost);

    printf("Block_Dim = %d, Grid_Dim = %d\n", Block_Dim, Grid_Dim);
    printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time
#endif

#ifdef CPU_PART
    // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
    // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

    hipEventRecord(start, 0); // use same timing
    // cudaEventSynchronize(start); // not needed

    kij_matrixmult(a, b, kij, m, n, p); // do calculation on host (NOTE: This computes the diff with GPU result.)

    hipEventRecord(stop, 0); // instrument code to measue end time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    printf("Time to calculate kij results on CPU: %f ms.\n", elapsed_time_ms); // exec. time

    
#endif

#ifdef CHECK_PART
    
    #ifdef PRINT_PART
        printf("\n");
        printf("Matrix A:\n");
        print_matrix(a, m, n);
        printf("Matrix B:\n");
        print_matrix(b, n, p);
        
        #ifdef GPU_PART
        printf("GPU Result:\n");
        print_matrix(gpu, m, p);
        #endif
        
        #ifdef CPU_PART
        printf("CPU Result:\n");
        print_matrix(kij, m, p);
        #endif
    #endif

    #ifdef CPU_PART
        check_matrix(a, b, kij, m, n, p);
    #endif    

    #ifdef GPU_PART
        check_matrix(a, b, gpu, m, n, p);
    #endif 
#endif



    // -------------- clean up ---------------------------------------

    free(a);
    free(b);
    free(gpu);

#ifdef GPU_PART
    cudaFree(dev_a);
    cudaFree(dev_b);
    cudaFree(dev_c);
#endif

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

