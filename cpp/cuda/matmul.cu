
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>


#ifdef GPU_PART
    //#define GPU_Task1
    #define GPU_Task2
    //#define GPU_Task34
    
#endif

#define NO_CMD_INPUT
#ifdef NO_CMD_INPUT
    #define M        8192
    #define N        2048
    #define P        2048    
    #define BlockDim 32
    #define GridDim  (N / BlockDim)
    #define TILE_WIDTH 16
#endif

//#define CPU_PART
#define GPU_PART
#define CHECK_PART
//#define PRINT_PART

#define rand_max 32768
#define FP float

void print_matrix(FP* a, int m, int n)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            printf("%f ", a[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void init_matrix(FP* a, int m, int n)
{
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            a[i * n + j] = (FP)rand() / (FP)rand_max;
            //      a[i * p + j] = (FP) i+j; // may be helpful for debugging
        }
    }
}

__global__ void gpu_kernel_task1(FP* a, FP* b, FP* c, int m, int n, int p)
{
    float sum = 0.0f;

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < p)
    {
        for (int i = 0; i < n; ++i)
        {
            sum += a[row * n + i] * b[i * p + col];
        }
        c[row * p + col] = sum;
    }
}

__global__ void gpu_kernel_task2(FP* a, FP* b, FP* c, int m, int n, int p)
{
    __shared__ float sharedA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sharedB[TILE_WIDTH][TILE_WIDTH];
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;
    float v = 0.0;

    for (int i = 0; i < (int)(ceil((float)n / TILE_WIDTH)); i++)
    {
        if (i * TILE_WIDTH + tx < n && row < m)
            sharedA[ty][tx] = a[row * n + i * TILE_WIDTH + tx];
        else
            sharedA[ty][tx] = 0.0;

        if (i * TILE_WIDTH + ty < n && col < p)
            sharedB[ty][tx] = b[(i * TILE_WIDTH + ty) * p + col];
        else
            sharedB[ty][tx] = 0.0;
        __syncthreads();

        for (int j = 0; j < TILE_WIDTH; j++)
            v += sharedA[ty][j] * sharedB[j][tx];
        __syncthreads();
    }

    if (row < m && col < p)
        c[row * p + col] = v;
}

void kij_matrixmult(FP* a, FP* b, FP* kij, int m, int n, int p)
{
    for (int k = 0; k < n; k++)
    {
        for (int i = 0; i < m; i++)
        {
            FP r = a[i * n + k];
            for (int j = 0; j < p; j++)
            {
                kij[i * p + j] += r * b[k * p + j];
            }
        }
    }
}

void check_matrix(FP* a, FP* b, FP* gpu, int m, int n, int p)
{
    FP* kij = (FP*)malloc(m * p * sizeof(FP)); // results from CPU
    memset(kij, 0, m * p * sizeof(FP));

    kij_matrixmult(a, b, kij, m, n, p);

    // diff on result matrix
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < p; j++)
        {
            gpu[i * p + j] -= kij[i * p + j];
        }
    }

    double error, suma, sumb, sumc;
    suma = 0.; sumb = 0; sumc = 0;
    
    // suma, m * n
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < n; j++)
        {
            suma += a[i * n + j] * a[i * n + j];
        }
    }

    // sumb, n * p
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < p; j++)
        {
            sumb += b[i * p + j] * b[i * p + j];
        }
    }
    
    // sumc, m * p
    for (int i = 0; i < m; i++)
    {
        for (int j = 0; j < p; j++)
        {
            sumc += gpu[i * p + j] * gpu[i * p + j];
        }
    }

    suma = sqrt(suma);
    sumb = sqrt(sumb);
    sumc = sqrt(sumc);
    error = sumc / (sqrt(m*p) * suma * sumb);
    printf("Scaled error between GPU and CPU: %f\n", error);
}

int main(int argc, char* argv[])
{
    int Grid_Dim = 1; //Grid dimension, x and y, square
    int Block_Dim = 1; //Block dimension, x and y, square
    int m, n, p; // matrix dimension
    FP* a, *b, *gpu;

    hipEvent_t start, stop; // using cuda events to measure time
    float elapsed_time_ms; // which is applicable for asynchronous code also
    hipError_t errorcode;

#ifdef NO_CMD_INPUT    
    m = M;
    n = N;
    p = P;
    Block_Dim = BlockDim;
    Grid_Dim = GridDim;
#else
    if (argc != 4)
    {
        printf("Usage: matmul <matrix dim> <block dim> <grid dim>\n");
        exit(-1);
    }
    n = atoi(argv[1]);
    Block_Dim = atoi(argv[2]); // Square block
    Grid_Dim = atoi(argv[3]); // Square grid
#endif

    a = (FP*)malloc(m * n * sizeof(FP)); // dynamically allocated memory for arrays on host
    b = (FP*)malloc(n * p * sizeof(FP));
    gpu = (FP*)malloc(m * p * sizeof(FP)); // results from GPU

    FP* kij = (FP*)malloc(m * p * sizeof(FP)); // results from CPU
    memset(kij, 0, m * p * sizeof(FP));

    srand(12345);
    init_matrix(a, m, n);
    init_matrix(b, n, p);

    hipEventCreate(&start); // instrument code to measure start time
    hipEventCreate(&stop);

#ifdef GPU_PART
    int gpucount = 0; // Count of available GPUs
    int gpunum = 0; // Device number to use
    FP* dev_a, * dev_b, * dev_c;

    // --------------------SET PARAMETERS AND DATA -----------------------

    errorcode = hipGetDeviceCount(&gpucount);
    if (errorcode == hipErrorNoDevice)
    {
        printf("No GPUs are visible\n");
        exit(-1);
    }
    else
    {
        printf("Device count = %d\n", gpucount);
    }

    if (Block_Dim * Block_Dim > 1024)
    {
        printf("Error, too many threads in block\n");
        exit(-1);
    }
    
    if (Grid_Dim * Block_Dim < n)
    {
        printf("Error, number of threads in x/y dimensions less than number of array elements\n");
        exit(-1);
    }

    hipSetDevice(gpunum);
    printf("Using device %d\n", gpunum);

    hipMalloc((void**)&dev_a, m * n * sizeof(FP)); // allocate memory on device
    hipMalloc((void**)&dev_b, n * p * sizeof(FP));
    hipMalloc((void**)&dev_c, m * p * sizeof(FP));

    hipMemcpy(dev_a, a, m * n * sizeof(FP), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * p * sizeof(FP), hipMemcpyHostToDevice);

    hipEventRecord(start, 0);
    // cudaEventSynchronize(start); // not needed

#ifdef GPU_Task1
    dim3 DimGrid((p - 1) / BlockDim + 1, (m - 1) / BlockDim + 1, 1);
    dim3 DimBlock(BlockDim, BlockDim, 1);
    gpu_kernel_task1 <<<DimGrid, DimBlock >>> (dev_a, dev_b, dev_c, m, n, p);
#endif    

#ifdef GPU_Task2
    dim3 DimGrid(ceil(p / BlockDim), ceil(m / BlockDim), 1);
    dim3 DimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    gpu_kernel_task2 << < dimGrid, dimBlock >> > (dev_a, dev_b, dev_c, m, n, p, TILE_WIDTH);
#endif

    hipEventRecord(stop, 0); // instrument code to measure end time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time_ms, start, stop);

    hipMemcpy(gpu, dev_c, m * p * sizeof(FP), hipMemcpyDeviceToHost);

    printf("Matrix Dimension = %d\n", n);
    printf("Block_Dim = %d, Grid_Dim = %d\n", Block_Dim, Grid_Dim);

    printf("Time to calculate results on GPU: %f ms.\n", elapsed_time_ms); // exec. time
#endif

#ifdef CPU_PART
    // ------------- COMPUTATION DONE ON HOST CPU ----------------------------
    // DEBUGGING USE ONLY (AND FOR LIMITED NUMBERS OF TIMING RUNS)

    cudaEventRecord(start, 0); // use same timing
    // cudaEventSynchronize(start); // not needed

    kij_matrixmult(a, b, kij, m, n, p); // do calculation on host (NOTE: This computes the diff with GPU result.)

    cudaEventRecord(stop, 0); // instrument code to measue end time
    cudaEventSynchronize(stop);
    cudaEventElapsedTime(&elapsed_time_ms, start, stop);

    printf("Time to calculate kij results on CPU: %f ms.\n", elapsed_time_ms); // exec. time
#endif

#ifdef CHECK_PART
    check_matrix(a, b, gpu, m, n, p);
#endif

#ifdef PRINT_PART
    print_matrix(a, m, n);
    print_matrix(b, n, p);
    print_matrix(kij, m, p);
#endif

    // -------------- clean up ---------------------------------------

    free(a);
    free(b);
    free(gpu);

#ifdef GPU_PART
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
#endif

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

